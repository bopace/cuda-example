﻿//****************************************************************************80
//
//  file name:
//
//    raise_to_power.cu
//
//  licensing:
//
//    this code is distributed under the mit license.
//
//  author:
//    bo pace
//
//  reference:
//    based on https://developer.nvidia.com/blog/even-easier-introduction-cuda/
//    an article by mark harris of nvidia

#include "hip/hip_runtime.h"


#include <iostream>
#include <math.h>

//****************************************************************************80
//
//  description:
//
//    cuda kernal function. raises the elements of one array to the power of the
//    elements of another array
//
//  last modified:
//
//    17 april 2021
//
//  input:
//
//	  int n - the length of the arrays
//    float* arr1 - pointer to array of bases
//	  float* arr2 - pointer to array of exponents
//
__global__
void raise_to_power(int n, float* arr1, float* arr2)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	{
		arr2[i] = pow(arr1[i], arr2[i]);
	}
}

int main(void)
{
	int arr_size = 1 << 20; // 1 million

	// allocate unified memory -- accessible from cpu or gpu
	float* arr1, * arr2;
	hipMallocManaged(&arr1, arr_size * sizeof(float));
	hipMallocManaged(&arr2, arr_size * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < arr_size; i++)
	{
		arr1[i] = 3.0f;
		arr2[i] = 2.0f;
	}

	int blockSize = 256;
	int numBlocks = (arr_size + blockSize - 1) / blockSize;
	raise_to_power <<<numBlocks, blockSize>>> (arr_size, arr1, arr2);

	// wait for gpu to finish before accessing on host
	hipDeviceSynchronize();

	// check for errors (all values should be 9.0f)
	float maxError = 0.0f;
	for (int i = 0; i < arr_size; i++)
	{
		maxError = fmax(maxError, fabs(arr2[i] - 9.0f));
	}

	std::cout << "Max error: " << maxError << '\n';

	// free memory
	hipFree(arr1);
	hipFree(arr2);

	return 0;
}
